#include "hip/hip_runtime.h"
#include <boost/test/utils/basic_cstring/basic_cstring_fwd.hpp>
#include <cstddef>
#include <cstdlib>
#include <iterator>
#include <vector>
#define NO_CPP11

#include <iostream>
#include <moderngpu.cuh>
#include <queue>

#include "pathway/GPU-solver.hpp"
#include "pathway/GPU-kernel.cuh"
#include "utils.hpp"
#include <fmt/core.h>
#include <memory>

bool debug = false;

using namespace mgpu;

using thrust::raw_pointer_cast;

int div_up(int x, int y) { return (x-1) / y + 1; }

struct RoutingData {
};

struct DeviceData {
    // store the structure of the grid graph
    MGPU_MEM(uint8_t) graph;

    // store open list + close list
    MGPU_MEM(node_t) nodes;
    MGPU_MEM(int) nodeSize;

    // hash table for `nodes'
    MGPU_MEM(uint32_t) hash;
    // define the modules of sub hash table (not required in pathway finding
    // MGPU_MEM(uint32_t) modules;

    // store open list
    MGPU_MEM(heap_t) openList;
    // store the size for each heap
    MGPU_MEM(int) heapSize;
    MGPU_MEM(int) heapBeginIndex;

    // element waiting to be sorted
    MGPU_MEM(sort_t) sortList;
    // value for sortList, representing the its parents
    MGPU_MEM(uint32_t) prevList;
    // size of the preceding array
    MGPU_MEM(int) sortListSize;

    MGPU_MEM(sort_t) sortList2;
    MGPU_MEM(uint32_t) prevList2;
    MGPU_MEM(int) sortListSize2;

    MGPU_MEM(heap_t) heapInsertList;
    MGPU_MEM(int) heapInsertSize;

    // current shortest distance (a float)
    MGPU_MEM(uint32_t) optimalDistance;
    // store the result return by the GPU
    MGPU_MEM(heap_t) optimalNodes;
    // store the size for optimalNodes
    MGPU_MEM(int) optimalNodesSize;

    MGPU_MEM(uint32_t) lastAddr;
    MGPU_MEM(uint32_t) answerList;
    MGPU_MEM(int) answerSize;

    ContextPtr context;
  device_vector<unsigned long long> bits;
  device_vector<bool> srcs;
  device_vector<bool> prevDirs;
  device_vector<bool> guides;
  device_vector<bool> zDirs;
  device_vector<int> xCoords;
  device_vector<int> yCoords;
  device_vector<int> zCoords;
  device_vector<int> zHeights;
  device_vector<frUInt4> path_widths;
  device_vector<int> via2ViaForbOverlapLen;
  device_vector<int> via2viaForbLen;
  device_vector<int> viaForbiTurnLen;
  device_vector<forBiddenRange_t *> overlap_addr;
  device_vector<forBiddenRange_t *> len_addr;
  device_vector<forBiddenRange_t *> turnlen_addr;
  forBiddenRange_t halfViaEncArea;
  int forBiddenRange_layerNum;
};


GPUPathwaySolver::GPUPathwaySolver()
{
    d = new DeviceData();
}

GPUPathwaySolver::~GPUPathwaySolver()
{
    // vector<node_t> nodes;
    // vector<uint32_t> hash;
    // d->nodes->ToHost(nodes, d->nodeSize->Value());
    // d->hash->ToHost(hash, p->size());
    // for (;;) {
    //     cout << "(x, y): ";
    //     int x, y;
    //     int px, py;
    //     cin >> x >> y;
    //     int nodeID = p->toID(x, y);
    //     int hashValue = hash[nodeID];
    //     int prevID = nodes[nodes[hashValue].prev].nodeID;
    //     p->toXY(prevID, &px, &py);
    //     std::cout << "fValue: " << nodes[hashValue].fValue << endl
    //               << "gValue: " << nodes[hashValue].gValue << endl
    //               << "prev: " << px << ", " << py << endl << endl;;
    // }

  /*
  for (size_t i = 0; i < d->forBiddenRange_layerNum; ++i) {
    auto ranges = d->overlap_addr[i];
    for (size_t j = 0; j < 8; ++j) {
      auto range = ranges + j;
      if (range->size > 0) {
        hipFree(range->data);
      }
      hipFree(range);
    }
  }
  */
    delete d;
}

forBiddenRange_t GPUPathwaySolver::vectorPairCpy(
    vector<std::pair<frCoord, frCoord>> const &hostData) {
  forBiddenRange_t cuVecPair;
  auto const payload_size = hostData.size();
  cuVecPair.size = payload_size;
  if (payload_size == 0) {
    cuVecPair.data = nullptr;
  } else {
    thrust::pair<frCoord, frCoord> *payload_ptr = nullptr;
    hipMalloc(&payload_ptr, sizeof(thrust::pair<frCoord, frCoord>) * payload_size);
    cuVecPair.data = payload_ptr;
    vector<thrust::pair<frCoord, frCoord>> thrust_pair_vector;
    for (auto const &j: hostData) {
      thrust::pair<frCoord, frCoord> device_pair(j);
      thrust_pair_vector.push_back(device_pair);
    }
    hipMemcpy(payload_ptr, thrust_pair_vector.data(), sizeof(thrust::pair<frCoord, frCoord>) * payload_size, hipMemcpyHostToDevice);
  }
  return cuVecPair;
}

void GPUPathwaySolver::forBiddenRangesDataCpy(
    device_vector<forBiddenRange_t *> &dest, 
    vector<vector<vector<std::pair<frCoord, frCoord>>>> const &hostData) {
    int const forBiddenRange_layerNum = (int) hostData.size();
    size_t const dirNum = hostData[0].size();
    hipMemcpyToSymbol(HIP_SYMBOL(d_forBiddenRange_layerNum), &forBiddenRange_layerNum, sizeof(int));
    for (auto const &hostFbRanges: hostData) {
      forBiddenRange_t deviceFbRanges[8];
      for (int i = 0; i < dirNum; ++i) {
        deviceFbRanges[i] = vectorPairCpy(hostFbRanges[i]);
      }
      forBiddenRange_t *ranges_ptr = nullptr;
      hipMalloc(&ranges_ptr, sizeof(forBiddenRange_t) * dirNum);
      hipMemcpy(ranges_ptr, deviceFbRanges, sizeof(forBiddenRange_t) * dirNum, hipMemcpyHostToDevice);
      dest.push_back(ranges_ptr);
    }
}

int GPUPathwaySolver::gpuKnows(int x, int y, int z) {
  int res = false;
  device_vector<int> resvec;
  resvec.push_back(0);
  auto vec_data_ptr = thrust::raw_pointer_cast(&d->zDirs[0]);
  auto resvec_ptr = thrust::raw_pointer_cast(&resvec[0]);
  read_bool_vec<<<1, 1>>>(resvec_ptr, x, y, z);
  res = resvec[0];
  return res;
}
bool GPUPathwaySolver::isEx(int x, int y, int z, frDirEnum dir, 
    frDirEnum lastdir) {
  bool res = false;
  device_vector<bool> resvec;
  resvec.push_back(false);
  auto res_ptr = thrust::raw_pointer_cast(&resvec[0]);
  test_isex<<<1, 1>>>(res_ptr, x, y, z, dir, lastdir);
  res = resvec[0];
  return res;
}

bool GPUPathwaySolver::testhasEdge(int x, int y, int z, frDirEnum dir) {
  bool res = false;
  device_vector<bool> resvec;
  resvec.push_back(false);
  auto res_ptr = thrust::raw_pointer_cast(&resvec[0]);
  test_isSrc<<<1, 1>>>(res_ptr, x, y, z);
  res = resvec[0];
  return res;
}

frDirEnum GPUPathwaySolver::testDir(int x, int y, int z) {
  frDirEnum res = frDirEnum::UNKNOWN;
  device_vector<frDirEnum> resvec;
  resvec.push_back(res);
  auto res_ptr = thrust::raw_pointer_cast(&resvec[0]);
  test_Dir<<<1, 1>>>(res_ptr, x, y, z);
  res = resvec[0];
  return res;
}


void GPUPathwaySolver::test_reverse(frMIdx &x, frMIdx &y, frMIdx &z, frDirEnum &dir) {
  device_vector<frDirEnum> dir_vec;
  dir_vec.push_back(dir);
  device_vector<frMIdx> idx_vec;
  idx_vec.push_back(x);
  idx_vec.push_back(y);
  idx_vec.push_back(z);
  auto dir_ptr = thrust::raw_pointer_cast(&dir_vec[0]);
  auto x_ptr = thrust::raw_pointer_cast(&idx_vec[0]);
  auto y_ptr = x_ptr + 1; // thrust::raw_pointer_cast(&idx_vec[1]);
  auto z_ptr = y_ptr + 1; // thrust::raw_pointer_cast(&idx_vec[2]);
  test_cuReverse<<<1, 1>>>(x_ptr, y_ptr, z_ptr, dir_ptr, x, y, z);
  x = idx_vec[0];
  y = idx_vec[1];
  z = idx_vec[2];
  dir = dir_vec[0];
}

cuWavefrontGrid GPUPathwaySolver::test_expand(frDirEnum dir, cuWavefrontGrid &grid, 
    const FlexMazeIdx &dstMazeIdx1, const FlexMazeIdx &dstMazeIdx2, 
    const frPoint &centerPt) {
  /*
  device_vector<cuWavefrontGrid> currgrid_vec;
  currgrid_vec.push_back(grid);
  device_vector<FlexMazeIdx> d_idx;
  d_idx.push_back(dstMazeIdx1);
  d_idx.push_back(dstMazeIdx2);
  device_vector<frPoint> ctrPt_vec;
  ctrPt_vec.push_back(centerPt);
  auto grid_ptr = raw_pointer_cast(&currgrid_vec[0]);
  auto center_ptr = raw_pointer_cast(&ctrPt_vec[0]);
  auto src_ptr = raw_pointer_cast(&d_idx[0]);
  auto dst1_ptr = src_ptr + 1;
  auto dst2_ptr = src_ptr + 2;
  device_vector<cuWavefrontGrid> resvec;
  resvec.push_back(0);
  auto res_ptr = raw_pointer_cast(&resvec[0]);
  test_cuexpand<<<1, 1>>>(res_ptr, grid_ptr, dir, dst1_ptr, dst2_ptr, center_ptr);
  auto res = resvec[0];
  */
  auto res = cuWavefrontGrid(grid);
  return res;
}

frCost GPUPathwaySolver::test_npCost(frDirEnum dir, cuWavefrontGrid &grid) {
  device_vector<frCost> resvec;
  resvec.push_back(0);
  auto res_ptr = raw_pointer_cast(&resvec[0]);
  test_getNCost_obj<<<1, 1>>>(res_ptr, dir, grid);
  auto res = resvec[0];
  return res;
}

frCoord GPUPathwaySolver::dtest_half(frMIdx z, bool f) {
  device_vector<frCoord> res;
  res.push_back(0);
  auto res_ptr = raw_pointer_cast(&res[0]);
  test_halfviaenc<<<1, 1>>>(res_ptr, z, f);
  auto hres = res[0];
  return hres;
}


void GPUPathwaySolver::printDeviceOverlapInfo(void){
  test_print_device_overlap_info<<<1, 1>>>();
}

frCost GPUPathwaySolver::test_npCost(frDirEnum dir, 
int xIn, int yIn, int zIn, frCoord layerPathAreaIn, 
          frCoord vLengthXIn, frCoord vLengthYIn,
          bool prevViaUpIn, frCoord tLengthIn,
          frCoord distIn, frCost pathCostIn, frCost costIn, 
          unsigned int backTraceBuffer
   ) {

  frCost res[1];
  frCost *d_res = nullptr;
  hipMalloc(&d_res, sizeof(frCost));
  test_getNCost<<<1, 1>>>(d_res, dir, xIn, yIn, zIn, layerPathAreaIn, vLengthXIn, vLengthYIn, 
      prevViaUpIn, tLengthIn, distIn, pathCostIn, costIn, backTraceBuffer);
  hipMemcpy(res, d_res, sizeof(frCost), hipMemcpyDeviceToHost);
  hipFree(d_res);
  return res[0];
}

frCost GPUPathwaySolver::test_estcost(FlexMazeIdx src, FlexMazeIdx dst1, FlexMazeIdx dst2, frDirEnum dir) {
  frCost res[1];
  frCost *d_res = nullptr;
  hipMalloc(&d_res, sizeof(frCost));
  dtest_estcost<<<1, 1>>>(d_res, src, dst1, dst2, dir);
  hipMemcpy(res, d_res, sizeof(frCost), hipMemcpyDeviceToHost);
  hipFree(d_res);
  return res[0];
}

void GPUPathwaySolver::initialize(const vector<unsigned long long> &bits, 
    const bovec &prevDirs, const bovec &srcs, 
        const bovec &guides, const bovec &zDirs, 
        const ivec &xCoords, const ivec &yCoords, const ivec &zCoords,
        const ivec &zHeights, const vector<frUInt4> &path_widths, 
        frUInt4 ggDRCCost, frUInt4 ggMarkerCost, 
        const ivec &via2ViaForbOverlapLen, const ivec &via2viaForbLen, 
        const ivec &viaForbiTurnLen, 
        bool drWorker_ava, int DRIter, int ripupMode, 
        int p_viaFOLen_size, int p_viaFLen_size, int p_viaFTLen_size, 
        vector<vector<vector<pair<frCoord, frCoord>>>> const &Via2ViaForbiddenOverlapLen, 
        vector<vector<vector<pair<frCoord, frCoord>>>> const &Via2ViaForbiddenLen,
        vector<vector<vector<pair<frCoord, frCoord>>>> const &ViaForbiddenTurnLen, 
        vector<std::pair<frCoord, frCoord>> const &halfViaEncArea_p, 
        std::string DBPROCESSNODE_p, frLayerNum topLayerNum_p
        )
{
    hipDeviceSynchronize();
    hipDeviceReset();

    d->context = CreateCudaDevice(0);

    d->bits = bits;
    d->prevDirs = prevDirs;
    d->srcs = srcs;
    d->guides = guides;
    d->zDirs = zDirs;
    d->xCoords = xCoords;
    d->yCoords = yCoords;
    d->zCoords = zCoords;
    d->zHeights= zHeights;
    d->path_widths = path_widths;
    d->via2ViaForbOverlapLen = via2ViaForbOverlapLen;
    d->via2viaForbLen = via2viaForbLen;
    d->viaForbiTurnLen = viaForbiTurnLen;

    int x = xCoords.size();
    int y = yCoords.size();
    int z = zCoords.size();

    auto bits_ptr = raw_pointer_cast(&d->bits[0]);
    auto prevDirs_ptr = raw_pointer_cast(&d->prevDirs[0]);
    auto srcs_ptr = raw_pointer_cast(&d->srcs[0]);
    auto guides_ptr = raw_pointer_cast(&d->guides[0]);
    auto zdirs_ptr = raw_pointer_cast(&d->zDirs[0]);
    auto xCoords_ptr = raw_pointer_cast(&d->xCoords[0]);
    auto yCoords_ptr = raw_pointer_cast(&d->yCoords[0]);
    auto zCoords_ptr = raw_pointer_cast(&d->zCoords[0]);
    auto zHeights_ptr = raw_pointer_cast(&d->zHeights[0]);
    auto path_widths_ptr = raw_pointer_cast(&d->path_widths[0]);
    auto vfol_ptr = raw_pointer_cast(&d->via2ViaForbOverlapLen[0]);
    auto v2vfl_ptr = raw_pointer_cast(&d->via2viaForbLen[0]);
    auto vftl_ptr = raw_pointer_cast(&d->viaForbiTurnLen[0]);



    d->forBiddenRange_layerNum = (int) Via2ViaForbiddenOverlapLen.size();
    hipMemcpyToSymbol(HIP_SYMBOL(d_forBiddenRange_layerNum), &d->forBiddenRange_layerNum, sizeof(int));
    forBiddenRangesDataCpy(d->overlap_addr, Via2ViaForbiddenOverlapLen);
    forBiddenRangesDataCpy(d->len_addr, Via2ViaForbiddenLen);
    forBiddenRangesDataCpy(d->turnlen_addr, ViaForbiddenTurnLen);
    auto overlap_addr_ptr = raw_pointer_cast(&d->overlap_addr[0]);
    auto len_addr_ptr = raw_pointer_cast(&d->len_addr[0]);
    auto turnlen_addr_ptr = raw_pointer_cast(&d->turnlen_addr[0]);

    d->halfViaEncArea = vectorPairCpy(halfViaEncArea_p);
    hipMemcpyToSymbol(HIP_SYMBOL(halfViaEncArea), &d->halfViaEncArea, sizeof(forBiddenRange_t));

    auto const strSize = DBPROCESSNODE_p.length() + 1;
    char const *str = nullptr;
    hipMalloc(&str, sizeof(char) * strSize);
    hipMemcpyToSymbol(HIP_SYMBOL(DBPROCESSNODE), &str, sizeof(char *));

    hipMemcpyToSymbol(HIP_SYMBOL(topLayerNum), &topLayerNum_p, sizeof(frLayerNum));

    initializeDevicePointers(bits_ptr, prevDirs_ptr, srcs_ptr, guides_ptr, zdirs_ptr,
        xCoords_ptr, yCoords_ptr, zCoords_ptr, zHeights_ptr, x, y, z,
        path_widths_ptr, ggDRCCost, ggMarkerCost, 
        drWorker, DRIter, ripupMode, 
        p_viaFOLen_size, p_viaFLen_size, p_viaFTLen_size, 
        overlap_addr_ptr, len_addr_ptr, turnlen_addr_ptr);
    // new array and copy
    /*
       initializeCUDAConstantMemory(
       p->height(), p->width(), p->layer(), p->ex(), p->ey(), p->ez(), 
       (uint32_t)p->toID(p->ex(), p->ey(), p->ez()));

       d->graph = d->context->Malloc<uint8_t>(p->graph(), p->size());

    d->nodes = d->context->Malloc<node_t>(NODE_LIST_SIZE);
    d->nodeSize = d->context->Fill<int>(1, 1);

    d->hash = d->context->Fill<uint32_t>(p->size(), UINT32_MAX);

    d->openList = d->context->Malloc<heap_t>(OPEN_LIST_SIZE);
    d->heapSize = d->context->Fill<int>(NUM_TOTAL, 0);
    d->heapBeginIndex = d->context->Fill<int>(1, 0);

    d->sortList = d->context->Malloc<sort_t>(NUM_VALUE * 8);
    d->prevList = d->context->Malloc<uint32_t>(NUM_VALUE * 8);
    d->sortList2 = d->context->Malloc<sort_t>(NUM_VALUE * 8);
    d->prevList2 = d->context->Malloc<uint32_t>(NUM_VALUE * 8);
    d->sortListSize = d->context->Fill<int>(1, 0);
    d->sortListSize2 = d->context->Fill<int>(1, 0);

    d->heapInsertList = d->context->Malloc<heap_t>(NUM_VALUE * 8);
    d->heapInsertSize = d->context->Fill<int>(1, 0);

    d->optimalDistance = d->context->Fill<uint32_t>(1, UINT32_MAX);
    d->optimalNodes = d->context->Malloc<heap_t>(NUM_TOTAL);
    d->optimalNodesSize = d->context->Fill<int>(1, 0);

    d->lastAddr = d->context->Malloc<uint32_t>(1);
    d->answerList = d->context->Malloc<uint32_t>(ANSWER_LIST_SIZE);
    d->answerSize = d->context->Fill<int>(1, 0);

    kInitialize<<<1, 1>>>(
        *d->nodes,
        *d->hash,
        *d->openList,
        *d->heapSize,
        p->sx(),
        p->sy(), 
        p->sz()
    );
    dout << "\t\tGPU Initialization finishes" << endl;
        */
}

bool GPUPathwaySolver::solve()
{
    std::priority_queue< heap_t, vector<heap_t>, std::greater<heap_t> > pq;

    for (int round = 0; ;++round) {
        if (DEBUG_CONDITION) {
            vector<int> heapSize;
            d->heapSize->ToHost(heapSize, NUM_TOTAL);
            printf("\t\t\t Heapsize: %d of %d\n", heapSize[0], HEAP_CAPACITY);
        }

        // printf("\t\tRound %d\n", round); fflush(stdout);
        dprintf("\t\tRound %d: kExtractExpand\n", round);
        kExtractExpand<
            NUM_BLOCK, NUM_THREAD, VALUE_PER_THREAD, HEAP_CAPACITY> <<<
            NUM_BLOCK, NUM_THREAD>>>(
                *d->nodes,

                *d->graph,

                *d->openList,
                *d->heapSize,

                *d->optimalDistance,
                *d->optimalNodes,
                *d->optimalNodesSize,

                *d->sortList,
                *d->prevList,
                *d->sortListSize,

                // reset them BTW
                *d->heapBeginIndex,
                *d->heapInsertSize
            );
#ifdef KERNEL_LOG
        hipDeviceSynchronize();
#endif

        dprintf("\t\tRound %d: Fetch optimalNodesSize: ", round);
        int optimalNodesSize = d->optimalNodesSize->Value();
        dprintf("%d\n", optimalNodesSize);

        if (optimalNodesSize) {
            printf("\t\tRound %d: Found one solution\n", round);
            vector<heap_t> optimalNodes;
            d->optimalNodes->ToHost(optimalNodes, optimalNodesSize);

            uint32_t optimalDistance = d->optimalDistance->Value();
            dprintf("\t\tRound %d: Fetch optimalDistance: %.2f\n", round, reverseFlipFloat(optimalDistance));

            for (size_t i = 0; i != optimalNodes.size(); ++i) {
                dprintf("\t\t\t optimalNodes[%d]: %.3f\n", (int)i, optimalNodes[i].fValue);
                pq.push(optimalNodes[i]);
            }

            dprintf("\t\t\t pq.top(): %.3f\n", pq.top().fValue);
            if (flipFloat(pq.top().fValue) <= optimalDistance) {
                printf("\t\t\t Number of nodes expanded: %d\n", d->nodeSize->Value());
                m_optimalNodeAddr = pq.top().addr;
                m_optimalDistance = pq.top().fValue;
                dprintf("\t\t\t Optimal nodes address: %d\n", m_optimalNodeAddr);
                return true;
            }
        }

        dprintf("\t\tRound %d: Fetch sortListSize: ", round);
        int sortListSize = d->sortListSize->Value();
        dprintf("%d\n", sortListSize);
        // if (round % 2000 == 0) {
        //     printf("\t\tRound %d: Fetch sortListSize: %d\n", round, sortListSize);
        // }
        if (sortListSize == 0)
            return false;

        dprintf("\t\tRound %d: MergesortPairs\n", round);
        MergesortPairs(
            d->sortList->get(),
            d->prevList->get(),
            sortListSize,
            *d->context
        );

        dprintf("\t\tRound %d: kAssign\n", round);
        kAssign<NUM_THREAD><<<
            div_up(sortListSize, NUM_THREAD), NUM_THREAD>>> (
                *d->sortList,
                *d->prevList,
                sortListSize,

                *d->sortList2,
                *d->prevList2,
                *d->sortListSize2
            );
#ifdef KERNEL_LOG
        hipDeviceSynchronize();
#endif

        dprintf("\t\tRound %d: Fetch sortListSize2: ", round);
        int sortListSize2 = d->sortListSize2->Value();
        dprintf("%d\n", sortListSize2);
        // if (round % 2000 == 0) {
        //     printf("\t\tRound %d: Fetch sortListSize2: %d\n", round, sortListSize2);
        // }

        dprintf("\t\tRound %d: kDeduplicate\n", round);
        // printf("\t\tRound %d: nodeSize: %d\n", round, d->nodeSize->Value());
        kDeduplicate<NUM_THREAD> <<<
            div_up(sortListSize2, NUM_THREAD), NUM_THREAD>>> (
                *d->nodes,
                *d->nodeSize,

                *d->hash,

                *d->sortList2,
                *d->prevList2,
                sortListSize2,

                *d->heapInsertList,
                *d->heapInsertSize
            );
        // printf("\t\tRound %d: nodeSize: %d\n", round, d->nodeSize->Value());
#ifdef KERNEL_LOG
        hipDeviceSynchronize();
#endif

        dprintf("\t\tRound %d: kHeapInsert\n", round);
        kHeapInsert<
            NUM_BLOCK, NUM_THREAD, HEAP_CAPACITY> <<<
            NUM_BLOCK, NUM_THREAD>>> (
                *d->openList,
                *d->heapSize,
                *d->heapBeginIndex,

                *d->heapInsertList,
                *d->heapInsertSize,

                // reset them BTW
                *d->sortListSize,
                *d->sortListSize2,
                *d->optimalDistance,
                *d->optimalNodesSize
            );
#ifdef KERNEL_LOG
        hipDeviceSynchronize();
#endif
        dprintf("\t\tRound %d: Finished\n\n", round);
    }
}

void GPUPathwaySolver::getSolution(float *optimal, vector<int> *pathList)
{
    d->lastAddr->FromHost(&m_optimalNodeAddr, 1);
    kFetchAnswer<<<1, 1>>>(
        *d->nodes,

        *d->lastAddr,

        *d->answerList,
        *d->answerSize
    );

    int answerSize = d->answerSize->Value();

    vector<uint32_t> answerList;
    d->answerList->ToHost(answerList, answerSize);

    *optimal = m_optimalDistance;
    pathList->clear();
    pathList->reserve(answerSize);
    for (int i = answerSize-1; i >= 0; --i) {
        pathList->push_back((int)answerList[i]);
    }

}

bool GPUPathwaySolver::isPrime(uint32_t number)
{
    uint32_t upper = sqrt(number) + 1;
    assert(upper < number);

    for (uint32_t i = 2; i != upper; ++i)
        if (number % i == 0)
            return false;
    return true;
}

vector<uint32_t> GPUPathwaySolver::genRandomPrime(uint32_t maximum, int count)
{
    vector<uint32_t> result;
    int prepare = 3 * count;

    uint32_t now = maximum;
    while (prepare) {
        if (isPrime(now))
            result.push_back(now);
        now--;
    }

    std::random_shuffle(result.begin(), result.end());
    result.erase(result.begin() + count, result.end());

    for (int i = 0; i < count; ++i)
        dout << result[i] << " ";
    dout << endl;

    return result;
}
